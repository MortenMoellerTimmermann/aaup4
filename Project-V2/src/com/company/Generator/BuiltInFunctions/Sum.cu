#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#define N 50

__global__ void MatrixSum(int matrix[][N], int Sum[][N]){
    __shared__ int LocalSum[N];
    int x = blockIdx.x * blockDim.x + threadIdx.x; 

    for(int i = 0; i < N; i++){
        LocalSum[x] += matrix[x][i];
    }
    __syncthreads();
    Sum[0][x] = LocalSum[x];
}

int main(){
    int A[N][N];
    int Sum[N];
    int finalSum = 0;
    int (*pA)[N], (*pS)[N];

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
        {
            A[i][j] = rand() % 2;
        }
    }

    hipMalloc((void**)&pA, (N*N)*sizeof(int));
    hipMalloc((void**)&pS, N*sizeof(int));

    hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    MatrixSum<<<dimGrid,dimBlock>>>(pA,pS);

    hipMemcpy(&Sum, pS, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        finalSum += Sum[i];
    }
    
    printf("A = \n");
    for(int i = 0; i < N; i++){
        int placeholder = 0;
        for(int j = 0; j < N; j++)
        {
            placeholder += A[i][j];
            printf("%d ", A[i][j]);
        }
        printf("P:%d \n", placeholder);
    }

    printf("Final Sum = %d\n", finalSum);

    hipFree(pA); 
    hipFree(pS);

    return 0;
}