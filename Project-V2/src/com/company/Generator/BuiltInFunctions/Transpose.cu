#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#define N 10

__global__ void transpose(int transpose[][N], int matrix[][N], int matrixSize){
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y; 

    if(x >= matrixSize || y>= matrixSize)
        return;

    printf("%d \n", matrix[x][y]);
    //int from = x + y * matrixSize;
    //int to = y + x * matrixSize;

    transpose[y][x] = matrix[x][y];
}   

int main(){
    int A[N][N] = {{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1}};
    int T[N][N] = {{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0}};    

    int (*pA)[N], (*pT)[N];

    hipMalloc((void**)&pA, (N*N)*sizeof(int));
    hipMalloc((void**)&pT, (N*N)*sizeof(int));

    hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pT, T, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = 1;
    dim3 threadsPerBlock(N,N);
    transpose_per_element<<<numBlocks,threadsPerBlock>>>(pT,pA,N*N);

    hipMemcpy(T, pT, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

    int i, j; printf("A = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", A[i][j]);
        }
        printf("\n");
    }

    printf("T = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", T[i][j]);
        }
        printf("\n");
    }

    hipFree(pA); 
    hipFree(pT);

    printf("\n");

    return 0;
}