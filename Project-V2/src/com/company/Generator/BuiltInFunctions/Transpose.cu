#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#define N 10

__global__ void transpose(int transpose[][N], int matrix[][N], int matrixSize){
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y; 

    if(x >= matrixSize || y>= matrixSize)
        return;

    transpose[y][x] = matrix[x][y];
}   

int main(){
    int A[N][N] = {{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1,},{2,2,2,2,2,2,2,2,2,2},{3,3,3,3,3,3,3,3,3,3},{1,1,1,1,1,1,1,1,1,1}};
    int T[N][N] = {{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0},{0,0,0,0,0,0,0,0,0,0}};    

    int (*pA)[N], (*pT)[N];

    hipMalloc((void**)&pA, (N*N)*sizeof(int));
    hipMalloc((void**)&pT, (N*N)*sizeof(int));

    hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pT, T, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    transpose_per_element<<<dimGrid,dimBlock>>>(pT,pA,N*N);

    hipMemcpy(T, pT, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

    int i, j; printf("A = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", A[i][j]);
        }
        printf("\n");
    }

    printf("T = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", T[i][j]);
        }
        printf("\n");
    }

    hipFree(pA); 
    hipFree(pT);

    printf("\n");

    return 0;
}