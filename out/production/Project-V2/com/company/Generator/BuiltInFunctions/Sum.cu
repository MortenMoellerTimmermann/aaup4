
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#define N 10

__global__ void MatrixSum(int matrix[][N], int Sum[][N]){
    __shared__ int LocalSum[N];
    int x = blockIdx.x * blockDim.x + threadIdx.x; 

    for(int i = 0; i < N; i++){
        LocalSum[x] += matrix[x][i];
    }
    __syncthreads();
    Sum[0][x] = LocalSum[x];
}

int main(){
    int A[N][N];
    int Sum[N];
    int finalSum = 0;
    int (*pA)[N], (*pS)[N];

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
        {
            A[i][j] = rand() % 5;
        }
    }

    hipMalloc((void**)&pA, (N*N)*sizeof(int));
    hipMalloc((void**)&pS, N*sizeof(int));

    hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = 1;
    dim3 threadsPerBlock(N,N);
    MatrixSum<<<numBlocks,N>>>(pA,pS);

    hipMemcpy(&Sum, pS, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        finalSum += Sum[i];
    }
    
    printf("A = \n");
    for(int i = 0; i < N; i++){
        int placeholder = 0;
        for(int j = 0; j < N; j++)
        {
            placeholder += A[i][j];
            printf("%d ", A[i][j]);
        }
        printf("P:%d \n", placeholder);
    }

    printf("Final Sum = %d\n", finalSum);

    hipFree(pA); 
    hipFree(pS);

    return 0;
}